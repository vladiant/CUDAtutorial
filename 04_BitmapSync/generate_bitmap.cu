#include "hip/hip_runtime.h"


#define _USE_MATH_DEFINES
#include <math.h>

// assume that the bitmap is rectangular, stride is neglected

__global__ void kernel(unsigned char* src)
{
  __shared__ float temp[16][16];

  int x = threadIdx.x + blockIdx.x * blockDim.x;
  int y = threadIdx.y + blockIdx.y * blockDim.y;
  int offset = x + y * blockDim.x * gridDim.x;
  const float period = 128.0f;
  temp[threadIdx.x][threadIdx.y] = 255 * 
    (sinf(x*2.0f*M_PI/period) + 1.0f) *
    (sinf(y*2.0f*M_PI/period) + 1.0f) / 4.0f;

  // comment out line below to see image break
  __syncthreads();

  src[offset*3] = 0;
  src[offset*3+1] = temp[15-threadIdx.x][15-threadIdx.y];
  src[offset*3+2] = 0;
}

extern "C" void GenerateBitmap(unsigned char* dst, int dimension)
{
  int size = dimension * dimension * 3; // assume RGB
  hipError_t status;

  // allocate as much memory
  unsigned char* src;
  status = hipMalloc(&src, size);
  
  dim3 blocks(dimension/16, dimension/16);
  dim3 threads(16,16);
  kernel<<<blocks,threads>>>(src);

  hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
  hipFree(src);
}

